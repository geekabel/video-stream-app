#include <hip/hip_runtime.h>
#include <math.h>

extern "C" {
__global__ void edge_detection(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int gx = -input[(y-1) * width + (x-1)] - 2*input[y * width + (x-1)] - input[(y+1) * width + (x-1)]
                 + input[(y-1) * width + (x+1)] + 2*input[y * width + (x+1)] + input[(y+1) * width + (x+1)];
        int gy = -input[(y-1) * width + (x-1)] - 2*input[(y-1) * width + x] - input[(y-1) * width + (x+1)]
                 + input[(y+1) * width + (x-1)] + 2*input[(y+1) * width + x] + input[(y+1) * width + (x+1)];
        output[idx] = min(255, max(0, (int)sqrtf(gx * gx + gy * gy)));
    }
}

void edgeDetection(unsigned char* input, unsigned char* output, int width, int height) {
    unsigned char* d_input;
    unsigned char* d_output;
    int size = width * height * sizeof(unsigned char);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    edge_detection<<<gridSize, blockSize>>>(d_input, d_output, width, height);

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
}
